

#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <stdlib.h>
#include <iomanip>
#include <sstream>
#include <sys/time.h>

using namespace std;

__managed__ int possible_msg;

__global__
void evaluate( unsigned long long int ciphertext,
            unsigned long long int modulus, unsigned long long int* solution )
{
    // index = block index * number of threads per block + thread index
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    // stride  = number threads per block * number of block per grid
    int stride = blockDim.x * gridDim.x;

    // vector<unsigned long long int> ker_res = {0};

    for(unsigned long long int curr = index; curr < modulus; curr += stride)
    {
        // compute cube mode to encrypt the current value of modulus
        unsigned long long int result = curr % modulus;

        for(int index = 0; index < 2; index++) {
            result = (result * curr) % modulus;
        }

        // add the result to the device vector,
        // and add a count of 1 to the number of solution available
        if( result == ciphertext )
        {
            atomicAdd(&possible_msg, 1);
            solution[curr] = 1; // if correct index, mark that index
        }
    }
}

void print_result(unsigned long long int* host_result,
        unsigned long long int ciphertext, unsigned long long int modulus)
{
    // iterate over the vector on the host to print the marked indices
    for(unsigned long long int index = 0; index < modulus; index++)
    {
        if( host_result[index] == 1 )
        {
            printf("%lld^3 = %lld (mod %lld)\n", index, ciphertext, modulus);
        }
    }
}

int main(int argc, char* argv[])
{
    // read from command line
    unsigned long long int ciphertext = atoi(argv[1]); // value of c
    unsigned long long int modulus = atoi(argv[2]); // value of n

    // initialize parameters
    clock_t start, end;

    //start timing
    start = clock();

    size_t size = modulus * sizeof(unsigned long long int);

    // Allocate input vectors h_A in host memory
    unsigned long long int* host_vec = (unsigned long long int*)malloc(size);

    // Allocate vectors in device memory
    unsigned long long int* dev_vec;
    hipMalloc(&dev_vec, size);

    // Copy vectors from host memory to device memory
    hipMemcpy(dev_vec, host_vec, size, hipMemcpyHostToDevice);

    // initialize the count of possible values
    possible_msg = 0;

    // Run kernel on the GPU
    int blockSize = 256;
    int numBlocks = (modulus + blockSize - 1) / blockSize;
    evaluate<<<numBlocks, blockSize>>>(ciphertext, modulus, dev_vec);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();


    if(possible_msg == 0) {

        printf("No cube roots of %lld (mod %lld)\n", ciphertext, modulus);

    } else {

        // Copy result from device memory to host memory
        hipMemcpy(host_vec, dev_vec, size, hipMemcpyDeviceToHost);

        // print results
        printf("There are %d possible messages which encryption give %lld\n",
                                                        possible_msg, ciphertext);
        print_result(host_vec, ciphertext, modulus);
    }

    // get stop time
    end = clock();
    double time_taken = double(end - start) / double(CLOCKS_PER_SEC);
    cout << "Breaking RSA GPU version 1 computation time : " << fixed
         << time_taken;
    cout << " seconds\n" << endl;

    // Free memory
    hipFree(dev_vec);
    delete host_vec;
    return 0;
}
